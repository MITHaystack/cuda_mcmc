//
// Compile:
//
// $ nvcc list_gpus.cu -o list_gpus
//
//

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>


int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device Index %d, %s, Compute Capability %d.%d\n",
               device, deviceProp.name, deviceProp.major, deviceProp.minor);
    }    
}
